#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "Cuda_historgram.h"
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <opencv2/opencv.hpp>
#include <iostream>

#include "freshman.h"
using namespace cv;
//__global__ void Sum(int *sum, int size, int* index)
//
//{
//
//	atomicAdd(sum, 1);
//    __syncthreads();
//}

//
//int main(void)
//
//{
//
//	cuda_historgram  cuda_Hist;
//	int Sum=0;
//	double hist_plot[256];
//	for (int i = 0; i < 256; ++i) {
//			std::cout<<cuda_Hist.Hist[i]<<std::endl;
//			Sum+=cuda_Hist.Hist[i];
//            hist_plot[i]=cuda_Hist.Hist[i]/300;
//
//		}
//	std::cout<<Sum<<std::endl;
//	Mat histImage( 300, 256, CV_8UC3, Scalar( 0,0,0) );
//
//	// rows cols
//
//	for(int i=1;i<256;i++)
//	{
//		line(histImage,Point( i-1, 300 - cvRound(hist_plot[i-1])) ,
//                Point( i, 300 - cvRound(hist_plot[i])) ,
//                Scalar( 255, 0, 0), 1, 8,0);
//
//
//	}
//	imshow("hist",histImage);
//	waitKey(0);
//	return 0;
//}

int main(void)

{     initDevice(0);

	double iStart=cpuSecond();
	cuda_historgram_rgb  cuda_Hist_rgb;
	double iElaps=cpuSecond()-iStart;
	 printf(" Time elapsed %f sec\n",iElaps);
	int Sum=0;
	double hist_plot[NUM_BINS*NUM_PART];
	for (int i = 0; i < NUM_BINS*NUM_PART; ++i) {
			std::cout<<cuda_Hist_rgb.Hist[i]<<std::endl;
			Sum+=cuda_Hist_rgb.Hist[i];
            hist_plot[i]=cuda_Hist_rgb.Hist[i]/100;

		}
	std::cout<<Sum<<std::endl;
	Mat histImage(512, 1024, CV_8UC3, Scalar( 255,255,255) );

	// rows cols

	for(int i=1;i<NUM_BINS*NUM_PART;i++)
	{
	  if(i<NUM_BINS)
		 line(histImage,Point( i*4, 512 ),
                Point( i*4, 512 - cvRound(hist_plot[i])) ,
                Scalar( 255,0,0), 4, 8,0);
	  else if (i<2*NUM_BINS)
		line(histImage,Point(( i-NUM_BINS)*4, 512 ),
		                Point( (i-NUM_BINS)*4, 512- cvRound(hist_plot[i])) ,
		                Scalar( 0,255, 0), 4, 8,0);
	  else
		line(histImage,Point(( i-NUM_BINS*2)*4, 512),
		                Point( (i-NUM_BINS*2)*4, 512 - cvRound(hist_plot[i])) ,
		                Scalar( 0, 0, 255), 4, 8,0);

	}

	imshow("hist",histImage);
	waitKey(0);

	return 0;
}
