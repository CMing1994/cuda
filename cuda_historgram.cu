#include "hip/hip_runtime.h"
#include "Cuda_historgram.h"


__global__ void historgram_shared(uchar *a, int height, int width,
		unsigned int *Out_His) {

	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	// grid dimensions
	int nx = blockDim.x * gridDim.x;
	int ny = blockDim.y * gridDim.y;

	// linear thread index within 2D block
	int t = threadIdx.x + threadIdx.y * blockDim.x;

	// total threads in 2D block
	int nt = blockDim.x * blockDim.y;

	// linear block index within 2D grid
	int g = blockIdx.x + blockIdx.y * gridDim.x;

	__shared__ unsigned int  smem[NUM_BINS];
         if(t<256) smem[t]=0;
   __syncthreads();
   //every block has a histogram
   for (int col = x; col < width; col += nx)
       for (int row = y; row < height; row += ny) {
    	   unsigned int Gray_Bin=(unsigned int)a[row*width+col];
    	   atomicAdd(&smem[Gray_Bin], 1);
       }

	__syncthreads();

    //write partial histogram into the global mem
    Out_His+=g*256;
	if(t<256){Out_His[t]=smem[t];
    }

}
__global__ void histogram_final_accum(const unsigned int *in, int n, unsigned int *out)
{  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if(i<256)
  {

	  for (int j = 0; j < n; j++) {
		  out[i] +=in[i+j*256];
	}

  }

}

cuda_historgram::~cuda_historgram(){

}
cuda_historgram::cuda_historgram() {
	image_read A;

	size_t size = A.height * A.width * sizeof(uchar);
	hipMalloc((void **)&d_A, size);

	hipMalloc((void **)&End_Hist,256*sizeof(unsigned int));
	hipMemcpy(d_A, A.input, size,hipMemcpyHostToDevice);

	dim3 dimBlock1(32, 16);
	dim3 dimGrid1((A.width-1)/dimBlock1.x+1,(A.height-1)/dimBlock1.y+1);
    int n =dimGrid1.x*dimGrid1.y;
    hipMalloc((void **)&Out_His,n*256*sizeof(unsigned int));

	historgram_shared<<<dimGrid1,dimBlock1>>>(d_A,A.height,A.width,Out_His);
	dim3 dimBlock2(256);
	dim3 dimGrid2(n);
    histogram_final_accum<<<dimGrid2,dimBlock2>>>(Out_His,n,End_Hist);
    hipMemcpy(Hist,End_Hist,256*sizeof(unsigned int),hipMemcpyDeviceToHost);



    hipFree(Out_His);
    hipFree(End_Hist);

	hipFree(d_A);
}

__global__ void historgram_shared_rgb(uchar3 *a, int height, int width,
		unsigned int *Out_His)
{

	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	// grid dimensions
	int nx = blockDim.x * gridDim.x;
	int ny = blockDim.y * gridDim.y;

	// linear thread index within 2D block
	int t = threadIdx.x + threadIdx.y * blockDim.x;

	// total threads in 2D block
	int nt = blockDim.x * blockDim.y;

	// linear block index within 2D grid
	int g = blockIdx.x + blockIdx.y * gridDim.x;

    __shared__ unsigned int  smem[NUM_BINS*NUM_PART];

    for(int i=t;i<NUM_BINS*NUM_PART;i+=nt){
    	smem[i]=0;
    }


   __syncthreads();
   //every block has a histogram
   // if not every pixel was included in the grid
   for (int col = x; col < width; col += nx)
       for (int row = y; row < height; row += ny) {

//   int col = x;
//   int row=y;
//   if(col<width&&row<height)
   {       unsigned int r_Bin=(unsigned int)a[row*width+col].x;
    	   unsigned int g_Bin=(unsigned int)a[row*width+col].y;
    	   unsigned int b_Bin=(unsigned int)a[row*width+col].z;

    	   atomicAdd(&smem[r_Bin], 1);
    	   atomicAdd(&smem[1*NUM_BINS+g_Bin], 1);
    	   atomicAdd(&smem[2*NUM_BINS+b_Bin], 1);}
      }

	__syncthreads();

  //  write partial histogram into the global mem
    Out_His+=g*NUM_BINS*NUM_PART;
    for(int i=t;i<NUM_BINS;i+=nt)
    {
		Out_His[i+NUM_BINS*0]=smem[i+NUM_BINS*0];
		Out_His[i+NUM_BINS*1]=smem[i+NUM_BINS*1];
		Out_His[i+NUM_BINS*2]=smem[i+NUM_BINS*2];

	}


}
/// the histogram of total
__global__ void histogram_final_accum_rgb(const unsigned int *in, int n, unsigned int *out)
{  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if(i<3*NUM_BINS)
  {
       unsigned int total=0;
	  for (int j = 0; j < n; j++) {
		  total +=in[i+j*NUM_BINS*NUM_PART];
	}
   out[i]=total;
  }

}

cuda_historgram_rgb::~cuda_historgram_rgb(){

}
cuda_historgram_rgb::cuda_historgram_rgb() {
	image_read A;
	hipProfilerStart();
	size_t size = A.height * A.width * sizeof(uchar3);
	hipMalloc((void **)&d_A, size);
	hipMemcpy(d_A, A.input, size,hipMemcpyHostToDevice);
     // the size of the endhist


	dim3 dimBlock1(32, 32);
	dim3 dimGrid1((A.width-1)/dimBlock1.x+1,(A.height-1)/dimBlock1.y+1);
    int n =dimGrid1.x*dimGrid1.y;
    hipMalloc((void **)&Out_His,n*NUM_PART*NUM_BINS*sizeof(unsigned int));
    hipMalloc((void **)&End_Hist,NUM_PART*NUM_BINS*sizeof(unsigned int));

	historgram_shared_rgb<<<dimGrid1,dimBlock1>>>(d_A,A.height,A.width,Out_His);
	dim3 dimBlock2(NUM_BINS*NUM_PART);
	dim3 dimGrid2(n);
	hipFree(d_A);
    histogram_final_accum_rgb<<<dimGrid2,dimBlock2>>>(Out_His,n,End_Hist);
//    hipMemcpy(Hist,End_Hist,NUM_PART*NUM_BINS*sizeof(unsigned int),hipMemcpyDeviceToHost);

    hipMemcpy(Hist,End_Hist,NUM_PART*NUM_BINS*sizeof(unsigned int),hipMemcpyDeviceToHost);


    hipFree(Out_His);
    hipFree(End_Hist);
    hipProfilerStop();

}
